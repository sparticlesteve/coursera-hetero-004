#include "hip/hip_runtime.h"
#include    <wb.h>

// Error check
#define wbCheck(stmt) do {                                                    \
        hipError_t err = stmt;                                               \
        if (err != hipSuccess) {                                             \
            wbLog(ERROR, "Failed to run stmt ", #stmt);                       \
            wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));    \
            return -1;                                                        \
        }                                                                     \
    } while(0)

// Vector addition kernel
__global__ void vecAdd(float * in1, float * in2, float * out, int len) {
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if(i < len)
        out[i] = in1[i] + in2[i];
}

// Function for transferring a chunk asyncronously via a stream
//hipError_t copySegment(float* out, float* in, int n, hipMemcpyKind kind, hipStream_t stream)


int main(int argc, char ** argv) {
    wbArg_t args;
    int inputLength;
    float * hostInput1;
    float * hostInput2;
    float * hostOutput;
    float * deviceInput1;
    float * deviceInput2;
    float * deviceOutput;

    args = wbArg_read(argc, argv);

    // TODO: try pinned host memory
    wbTime_start(Generic, "Importing data and creating memory on host");
    hostInput1 = (float *) wbImport(wbArg_getInputFile(args, 0), &inputLength);
    hostInput2 = (float *) wbImport(wbArg_getInputFile(args, 1), &inputLength);
    hostOutput = (float *) malloc(inputLength * sizeof(float));
    wbTime_stop(Generic, "Importing data and creating memory on host");
    
    // Allocate memory on the device
    // Start with just one set of device arrays.
    // I should be able to do the sectioning on that directly
    int size = inputLength * sizeof(float);
    wbCheck( hipMalloc((void**)&deviceInput1, size) );
    wbCheck( hipMalloc((void**)&deviceInput2, size) );
    wbCheck( hipMalloc((void**)&deviceOutput, size) );
    
    // Create the streams.
    // Let's now do 2 streams.
    hipStream_t stream0, stream1;
    hipStreamCreate(&stream0);
    hipStreamCreate(&stream1);
    
    // Block and segment size
    // Start with one segment
    const int blockSize = 256;
    int segSize = inputLength/2;
    int gridSize = (segSize-1) / blockSize + 1;
    
    printf("InputLength %i, blockSize %i, segSize %i, gridSize %i\n",
           inputLength, blockSize, segSize, gridSize);

    // Asynchronous transfer of stream0
    wbCheck( hipMemcpyAsync(deviceInput1, hostInput1, segSize*sizeof(float), hipMemcpyHostToDevice, stream0) );
    wbCheck( hipMemcpyAsync(deviceInput2, hostInput2, segSize*sizeof(float), hipMemcpyHostToDevice, stream0) );
    
    // Asynchronous transfer of stream1
    wbCheck( hipMemcpyAsync(deviceInput1+segSize, hostInput1+segSize, segSize*sizeof(float), hipMemcpyHostToDevice, stream1) );
    wbCheck( hipMemcpyAsync(deviceInput2+segSize, hostInput2+segSize, segSize*sizeof(float), hipMemcpyHostToDevice, stream1) );

    // Perform computation
    vecAdd<<<gridSize, blockSize, 0, stream0>>>(deviceInput1, deviceInput2, deviceOutput, inputLength);
    vecAdd<<<gridSize, blockSize, 0, stream1>>>(deviceInput1+segSize, deviceInput2+segSize, deviceOutput+segSize, inputLength);
    
    // Asynchronous return transfer
    wbCheck( hipMemcpyAsync(hostOutput, deviceOutput, segSize*sizeof(float), hipMemcpyDeviceToHost, stream0) );
    wbCheck( hipMemcpyAsync(hostOutput+segSize, deviceOutput+segSize, segSize*sizeof(float), hipMemcpyDeviceToHost, stream1) );

    // Wait for remaining streams to finish
    wbCheck( hipDeviceSynchronize() );
    
    // Debugging result
    const int nDump = 5;
    printf("Results\n");
    for(int i = 0; i < nDump; ++i){
        printf("  %i %f %f %f\n", i, hostInput1[i], hostInput2[i], hostOutput[i]);
    }

    // Check solution
    wbSolution(args, hostOutput, inputLength);
    
    // Free device memory
    wbCheck( hipFree(deviceInput1) );
    wbCheck( hipFree(deviceInput2) );
    wbCheck( hipFree(deviceOutput) );

    // Free host memory
    free(hostInput1);
    free(hostInput2);
    free(hostOutput);

    return 0;
}
