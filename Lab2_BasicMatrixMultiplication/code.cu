#include "hip/hip_runtime.h"
#include <wb.h>

#define wbCheck(stmt)                                                          \
  do {                                                                         \
    hipError_t err = stmt;                                                    \
    if (err != hipSuccess) {                                                  \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                              \
      wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));           \
      return -1;                                                               \
    }                                                                          \
  } while (0)

// Compute C = A * B
__global__ void matrixMultiply(float *A, float *B, float *C, int numARows,
                               int numAColumns, int numBRows, int numBColumns,
                               int numCRows, int numCColumns) {
    //@@ Insert code to implement matrix multiplication here
    
    // Calculate my positions in the C matrix
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Boundary check
    if(row < numCRows && col < numCColumns){
        // My value at C[row][col] is given by the dot-product of A[row][] and B[][col]
        float tempC = 0;
        for(int i = 0; i < numAColumns; ++i){
            tempC += A[row*numAColumns + i] * B[col + i*numBColumns];
        }
    
        // Write back to global memory
        C[row*numCColumns + col] = tempC;
    }
}

int main(int argc, char **argv) {
    wbArg_t args;
    float *hostA; // The A matrix
    float *hostB; // The B matrix
    float *hostC; // The output C matrix
    float *deviceA;
    float *deviceB;
    float *deviceC;
    int numARows;    // number of rows in the matrix A
    int numAColumns; // number of columns in the matrix A
    int numBRows;    // number of rows in the matrix B
    int numBColumns; // number of columns in the matrix B
    int numCRows;    // number of rows in the matrix C (you have to set this)
    int numCColumns; // number of columns in the matrix C (you have to set this)

    args = wbArg_read(argc, argv);

    wbTime_start(Generic, "Importing data and creating memory on host");
    hostA =
        ( float * )wbImport(wbArg_getInputFile(args, 0), &numARows, &numAColumns);
    hostB =
        ( float * )wbImport(wbArg_getInputFile(args, 1), &numBRows, &numBColumns);
    
    //@@ Set numCRows and numCColumns
    numCRows = numARows;
    numCColumns = numBColumns;
    
    //@@ Allocate the hostC matrix
    hostC = new float[numCRows * numCColumns];
    
    wbTime_stop(Generic, "Importing data and creating memory on host");

    wbLog(TRACE, "The dimensions of A are ", numARows, " x ", numAColumns);
    wbLog(TRACE, "The dimensions of B are ", numBRows, " x ", numBColumns);

    wbTime_start(GPU, "Allocating GPU memory.");
    
    //@@ Allocate GPU memory here
    int sizeA = numARows * numAColumns * sizeof(float);
    int sizeB = numBRows * numBColumns * sizeof(float);
    int sizeC = numCRows * numCColumns * sizeof(float);
    wbCheck( hipMalloc((void**)&deviceA, sizeA) );
    wbCheck( hipMalloc((void**)&deviceB, sizeB) );
    wbCheck( hipMalloc((void**)&deviceC, sizeC) );

    wbTime_stop(GPU, "Allocating GPU memory.");

    wbTime_start(GPU, "Copying input memory to the GPU.");
    
    //@@ Copy memory to the GPU here
    wbCheck( hipMemcpy(deviceA, hostA, sizeA, hipMemcpyHostToDevice) );
    wbCheck( hipMemcpy(deviceB, hostB, sizeB, hipMemcpyHostToDevice) );

    wbTime_stop(GPU, "Copying input memory to the GPU.");

    //@@ Initialize the grid and block dimensions here
    
    // Choose a block size
    const int tileWidth = 16;
    dim3 blockSize(tileWidth, tileWidth, 1);
    // Choose grid size based on the block size and dataset
    dim3 gridSize((numCColumns-1)/tileWidth + 1, (numCRows-1)/tileWidth + 1, 1);

    wbTime_start(Compute, "Performing CUDA computation");
    
    //@@ Launch the GPU Kernel here
    matrixMultiply<<<gridSize, blockSize>>>(deviceA, deviceB, deviceC,
                                            numARows, numAColumns, 
                                            numBRows, numBColumns, 
                                            numCRows, numCColumns);

    hipDeviceSynchronize();
    wbTime_stop(Compute, "Performing CUDA computation");

    wbTime_start(Copy, "Copying output memory to the CPU");
    
    //@@ Copy the GPU memory back to the CPU here
    wbCheck( hipMemcpy(hostC, deviceC, sizeC, hipMemcpyDeviceToHost) );

    wbTime_stop(Copy, "Copying output memory to the CPU");

    wbTime_start(GPU, "Freeing GPU Memory");
    
    //@@ Free the GPU memory here
    wbCheck( hipFree(deviceA) );
    wbCheck( hipFree(deviceB) );
    wbCheck( hipFree(deviceC) );

    wbTime_stop(GPU, "Freeing GPU Memory");

    wbSolution(args, hostC, numCRows, numCColumns);

    free(hostA);
    free(hostB);
    free(hostC);

    return 0;
}

