#include "hip/hip_runtime.h"
// Histogram Equalization

#include    <wb.h>

#define HISTOGRAM_LENGTH 256

#define wbCheck(stmt) do {                                                    \
        hipError_t err = stmt;                                               \
        if (err != hipSuccess) {                                             \
            wbLog(ERROR, "Failed to run stmt ", #stmt);                       \
            wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));    \
            return -1;                                                        \
        }                                                                     \
    } while(0)


//-----------------------------------------------------------------------------
// Convert image array of float into uchar.
//-----------------------------------------------------------------------------
__global__ void convertToChar(float* input, unsigned char* output, int len)
{
    // Thread stride is total num of threads in the grid
    int stride = blockDim.x * gridDim.x;
    // Loop over my elements
    for(int i = blockIdx.x*blockDim.x + threadIdx.x; i < len; i += stride)
        output[i] = (unsigned char) (255 * input[i]);
}

//-----------------------------------------------------------------------------
// Convert RGB image array into grayscale.
//-----------------------------------------------------------------------------
__global__ void convertToGrayScale(unsigned char* input, unsigned char* output,
                                   int numPixels)
{
    // Pixel index
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if(i < numPixels){
        // Combine red, blue, green to produce gray
        int inIdx = 3*i;
        unsigned char r = input[inIdx];
        unsigned char g = input[inIdx+1];
        unsigned char b = input[inIdx+2];
        // These factors were provided in the assignment
        output[i] = (unsigned char) (0.21*r + 0.71*g + 0.07*b);
    }
}

//-----------------------------------------------------------------------------
// Calculate histogram from grayscale data
//-----------------------------------------------------------------------------
__global__ void computeHistogram(unsigned char* input, unsigned int* histo,
                                 int numPixels)
{
    // Shared copy of histogram for each thread block
    __shared__ unsigned int sharedHist[HISTOGRAM_LENGTH];
    // Initialize histogram
    if(threadIdx.x < HISTOGRAM_LENGTH) sharedHist[threadIdx.x] = 0;
    __syncthreads();
    // Thread stride is total num of threads in the grid
    int stride = blockDim.x * gridDim.x;
    // Loop over my elements
    for(int i = blockIdx.x*blockDim.x + threadIdx.x;
        i < numPixels; i += stride){
        atomicAdd(&sharedHist[input[i]], 1);
    }
    // Wait for the whole block to finish
    __syncthreads();
    // Accumulate results into output
    if(threadIdx.x < HISTOGRAM_LENGTH)
        atomicAdd(&histo[threadIdx.x], sharedHist[threadIdx.x]);
}

//-----------------------------------------------------------------------------
// Single-block scan kernel for calculating histogram CDF
//-----------------------------------------------------------------------------
__global__ void computeCDF(unsigned int* histo, float* cdf, int numPixels)
{
    // Thread index in the block
    unsigned int t = threadIdx.x;
    // Shared memory
    __shared__ unsigned int shared[HISTOGRAM_LENGTH];
    // Load two elements into shared memory
    if(t < HISTOGRAM_LENGTH)
        shared[t] = histo[t];
    else
        shared[t] = 0;
    if(t + blockDim.x < HISTOGRAM_LENGTH)
        shared[t+blockDim.x] = histo[t+blockDim.x];
    else
        shared[t+blockDim.x] = 0;
    __syncthreads();
    // Downsweep
    for(unsigned int s = 1; s <= blockDim.x; s *= 2){
        int index = (t+1)*s*2 - 1;
        if(index < 2*blockDim.x){
            shared[index] += shared[index-s];
        }
        __syncthreads();
    }
    // Upsweep
    for(int s = blockDim.x/2; s > 0; s /= 2){
        int index = (t+1)*s*2 - 1;
        if(index+s < 2*blockDim.x){
            shared[index+s] += shared[index];
        }
        __syncthreads();
    }
    // Write to global output
    // Scale with probability factor here
    float factor = 1. / numPixels;
    if(t < HISTOGRAM_LENGTH)
        cdf[t] = shared[t]*factor;
    if(t+blockDim.x < HISTOGRAM_LENGTH)
        cdf[t+blockDim.x] = shared[t+blockDim.x]*factor;
}

//-----------------------------------------------------------------------------
// Histogram equalization kernel uses the CDF to scale every pixel channel.
//-----------------------------------------------------------------------------
__global__ void equalizeImage(unsigned char* image, float* cdf, int len)
{
    // Thread stride is total num of threads in the grid
    int stride = blockDim.x * gridDim.x;
    float cdfMin = cdf[0];
    // Loop over my elements
    for(int i = blockIdx.x*blockDim.x + threadIdx.x; i < len; i += stride){
        // get the uchar value at this element
        unsigned char val = image[i];
        // compute the corrected value
        int corVal = (int) ((cdf[val] - cdfMin)/(1. - cdfMin)*255.);
        corVal = min(corVal, 255);
        corVal = max(corVal, 0);
        image[i] = corVal;
    }
}

//-----------------------------------------------------------------------------
// Convert uchar image array back into float
//-----------------------------------------------------------------------------
__global__ void convertCharToFloat(unsigned char* input, float* output, int len)
{
    // Thread stride is total num of threads in the grid
    int stride = blockDim.x * gridDim.x;
    // Loop over my elements
    for(int i = blockIdx.x*blockDim.x + threadIdx.x; i < len; i += stride)
        output[i] = (float) (input[i]/255.);
}

//-----------------------------------------------------------------------------
// Main function
//-----------------------------------------------------------------------------
int main(int argc, char ** argv) {
    wbArg_t args;
    int imageWidth;
    int imageHeight;
    int imageChannels;
    wbImage_t inputImage;
    wbImage_t outputImage;
    const char * inputImageFile;
    float * hostInputImageData;
    float * hostOutputImageData;
    float * deviceInputImageData;
    float * deviceOutputImageData;
    unsigned char * deviceRGBData;
    unsigned char * deviceGrayData;
    unsigned int * deviceHist;
    float * deviceCDF;

    args = wbArg_read(argc, argv); /* parse the input arguments */

    inputImageFile = wbArg_getInputFile(args, 0);

    wbTime_start(Generic, "Importing data and creating memory on host");
    inputImage = wbImport(inputImageFile);
    imageWidth = wbImage_getWidth(inputImage);
    imageHeight = wbImage_getHeight(inputImage);
    imageChannels = wbImage_getChannels(inputImage);
    outputImage = wbImage_new(imageWidth, imageHeight, imageChannels);
    hostInputImageData = wbImage_getData(inputImage); // added
    hostOutputImageData = wbImage_getData(outputImage); // added
    wbTime_stop(Generic, "Importing data and creating memory on host");

    // Dump some information
    wbLog(INFO, "img width  ", imageWidth);
    wbLog(INFO, "img height ", imageHeight);
    wbLog(INFO, "img chans  ", imageChannels);

    // Allocate memory on device
    int numPixels = imageWidth*imageHeight;
    int imageLen = numPixels*imageChannels;
    wbTime_start(GPU, "Allocating GPU memory.");
    wbCheck( hipMalloc((void**)&deviceInputImageData, imageLen*sizeof(float)) );
    wbCheck( hipMalloc((void**)&deviceOutputImageData, imageLen*sizeof(float)) );
    wbCheck( hipMalloc((void**)&deviceRGBData, imageLen*sizeof(unsigned char)) );
    wbCheck( hipMalloc((void**)&deviceGrayData, numPixels*sizeof(unsigned char)) );
    wbCheck( hipMalloc((void**)&deviceHist, HISTOGRAM_LENGTH*sizeof(unsigned int)) );
    wbCheck( hipMalloc((void**)&deviceCDF, HISTOGRAM_LENGTH*sizeof(float)) );
    wbTime_stop(GPU, "Allocating GPU memory.");

    // Initializing histogram
    wbCheck( hipMemset(deviceHist, 0, HISTOGRAM_LENGTH*sizeof(unsigned int)) );

    // Transfer input data to device
    wbTime_start(GPU, "Copying input to GPU.");
    wbCheck( hipMemcpy(deviceInputImageData, hostInputImageData,
                        imageLen*sizeof(float), hipMemcpyHostToDevice) );
    wbTime_stop(GPU, "Copying input to GPU.");

    //-------------------------------------------------------------------------
    // Begin kernel computations
    //-------------------------------------------------------------------------
    wbTime_start(Compute, "Performing kernel computations.");

    // Debugging: dump out N input pixels
    /*const int nDump = 10;
    const int start = 0;
    float* h = hostInputImageData;
    printf("Input image data:");
    for(int i = start; i < start+nDump; ++i){
        int idx = 3*i;
        printf("  %i RGB %f, %f, %f\n", i, h[idx], h[idx+1], h[idx+2]);
    }*/

    // Convert image data to unsigned char. Strided kernel
    int blockSize1 = 512;
    int gridSize1 = ((imageLen-1) / blockSize1 + 1)/2;
    printf("Converting to uchar with %i blocks of %i threads\n",
           gridSize1, blockSize1);
    convertToChar<<<gridSize1, blockSize1>>>
        (deviceInputImageData, deviceRGBData,imageLen);

    // Debugging: dump the uchar
    /*unsigned char* hostRGBData = new unsigned char[imageLen];
    wbCheck( hipMemcpy(hostRGBData, deviceRGBData,
                        imageLen*sizeof(unsigned char),
                        hipMemcpyDeviceToHost) );
    printf("Input uchar RGB:\n");
    for(unsigned int i = start; i < start+nDump; ++i){
        int idx = 3*i;
        printf("  %i RGB %hhu, %hhu, %hhu\n", i,
               hostRGBData[idx], hostRGBData[idx+1], hostRGBData[idx+2]);
    }*/

    // Convert RGB to gray-scale
    int blockSize2 = 1024;
    int gridSize2 = (numPixels-1) / blockSize2 + 1;
    printf("Converting RGB to grayscale with %i blocks of %i threads\n",
           gridSize2, blockSize2);
    convertToGrayScale<<<gridSize2, blockSize2>>>
        (deviceRGBData, deviceGrayData, numPixels);

    // Debugging: dump the grayscale
    /*unsigned char * hostGrayData = new unsigned char[numPixels];
    wbCheck( hipMemcpy(hostGrayData, deviceGrayData,
                        numPixels*sizeof(unsigned char),
                        hipMemcpyDeviceToHost) );
    printf("Grayscale:\n");
    for(unsigned int i = start; i < start+nDump; ++i){
        printf("  %i Gray %hhu\n", i, hostGrayData[i]);
    }*/

    // Calculate the histogram. Strided kernel allows any block/grid size.
    int blockSize3 = 512;
    int gridSize3 = gridSize2/5; // each thread will do ~10 elements
    printf("Calculating histogram with %i blocks of %i threads\n",
           gridSize3, blockSize3);
    computeHistogram<<<gridSize3, blockSize3>>>
        (deviceGrayData, deviceHist, numPixels);

    // Debugging: copy histogram and dump
    /*unsigned int hostHist[HISTOGRAM_LENGTH];
    wbCheck( hipMemcpy(hostHist, deviceHist,
                        HISTOGRAM_LENGTH*sizeof(unsigned int),
                        hipMemcpyDeviceToHost) );
    printf("Histogram result:\n");
    for(unsigned int i = 0; i < HISTOGRAM_LENGTH; ++i){
        printf("  %i %i\n", i, hostHist[i]);
    }*/

    // Calculate the CDF via scan
    int blockSize4 = HISTOGRAM_LENGTH/2;
    int gridSize4 = 1;
    printf("Calculating CDF with %i blocks of %i threads\n",
           gridSize4, blockSize4);
    computeCDF<<<gridSize4, blockSize4>>>(deviceHist, deviceCDF, numPixels);

    // Debugging: copy CDF and dump
    /*float hostCDF[HISTOGRAM_LENGTH];
    wbCheck( hipMemcpy(hostCDF, deviceCDF, HISTOGRAM_LENGTH*sizeof(float),
                        hipMemcpyDeviceToHost) );
    printf("CDF result:\n");
    for(unsigned int i = 0; i < HISTOGRAM_LENGTH; ++i){
        printf("  %i %f\n", i, hostCDF[i]);
    }*/

    // Equalize the RGB image data using the CDF - strided kernel
    int blockSize5 = blockSize1;
    int gridSize5 = gridSize1;
    printf("Equalizing the RGB with %i blocks of %i threads\n",
           gridSize5, blockSize5);
    equalizeImage<<<gridSize5, blockSize5>>>
        (deviceRGBData, deviceCDF, imageLen);

    // Debugging: output uchar RGB data
    /*wbCheck( hipMemcpy(hostRGBData, deviceRGBData,
                          imageLen*sizeof(unsigned char),
                          hipMemcpyDeviceToHost) );
    printf("Output uchar RGB:\n");
    for(unsigned int i = start; i < start+nDump; ++i){
        int idx = 3*i;
        printf("  %i RGB %hhu, %hhu, %hhu\n", i,
               hostRGBData[idx], hostRGBData[idx+1], hostRGBData[idx+2]);
    }*/

    // Convert uchar image data back to float - strided kernel
    int blockSize6 = blockSize1;
    int gridSize6 = gridSize1;
    printf("Converting to float with %i blocks of %i threads\n",
           gridSize6, blockSize6);
    convertCharToFloat<<<gridSize6, blockSize6>>>
        (deviceRGBData, deviceOutputImageData, imageLen);

    wbCheck( hipDeviceSynchronize() );

    // End kernel computations
    wbTime_stop(Compute, "Performing kernel computations.");

    // Copy output data back to host
    wbCheck( hipMemcpy(hostOutputImageData, deviceOutputImageData,
                        imageLen*sizeof(float), hipMemcpyDeviceToHost) );

    // Debugging: dump out N output pixels
    /*printf("Output image data:\n");
    h = hostOutputImageData;
    for(int i = start; i < start+nDump; ++i){
        int idx = 3*i;
        printf("  %i RGB %f, %f, %f\n", i, h[idx], h[idx+1], h[idx+2]);
    }*/

    // Check solution
    wbSolution(args, outputImage);

    // Free GPU memory
    wbTime_start(GPU, "Freeing GPU memory.");
    hipFree(deviceInputImageData);
    hipFree(deviceOutputImageData);
    hipFree(deviceRGBData);
    hipFree(deviceGrayData);
    hipFree(deviceHist);
    hipFree(deviceCDF);
    wbTime_stop(GPU, "Freeing GPU memory.");

    // Free host memory
    wbImage_delete(inputImage);
    wbImage_delete(outputImage);

    // Free debug memory
    //delete[] hostRGBData;
    //delete[] hostGrayData;

    return 0;
}
